// File: LJforceMD.cu
// -------------------
// File containing methods to implement the LJforce
// class.

// This code was specifically developed to illustrate concepts in the accompanying book:
// R. J. Sadus, "Molecular Simulation of Fluids: Theory, Algorithms, Object-Orientation,
// and Parallel Computing," 2nd Ed. (Elsevier, Amsterdam, 2023). It can be used freely for
// any not for profit purpose or academic research application. The code has been validated,
// but it would be nonetheless prudent to test it further before publishing any results.
// Check the book's website for any subsequent updates.

#include "cudaSimKernels.h"
#include "ljforceMD.h"
#include <iostream>
#include <stdlib.h>

using namespace std;

// Method: setForce
// Usage: setForce();
// ------------------
// The ljForce method uses a GPU kernel to calculate the force
// experienced by all num atoms due to pairwise interatomic interaction.
// The forces are calculated using the kernel for the Lennard-Jones (6-12)
// potential. This can be replaced by kernels for other potentials.
// The potential is truncated at a distance of rCut and
// long range corrections must be applied outside of the
// the method to obtain the full contributions to both
// energy and the virial.

void LJforce::setForce(int num, double length, double* potEnergy, double* virial) {

    int* kind_dev;               // atom type;
    double* r_dev;               // atom position vectors
    double *energy, *energy_dev; // potential energy;
    double *vir, *vir_dev;       // virial
    double* force_dev;           // forces
    double *sigma_dev, *epsilon_dev, *rCut_dev;
    double potE, virE;

    // Distribute the calculations among the threads
    int blockSize = numThreadsInBlock;
    int numBlocks = (num + blockSize - 1) / blockSize;

    // Reinitalise force array prior to kernel execution
    for (int i = 0; i < num; i++) {
        atoms[0]->f[3 * i] = 0;
        atoms[0]->f[3 * i + 1] = 0;
        atoms[0]->f[3 * i + 2] = 0;
    }

    // Allocate memory on the CPU side
    energy = (double*)malloc(numBlocks * sizeof(double));
    vir = (double*)malloc(numBlocks * sizeof(double));

    // Allocate memory on the GPU
    hipMalloc((void**)&r_dev, 3 * num * sizeof(double));
    hipMalloc((void**)&force_dev, 3 * num * sizeof(double));
    hipMalloc((void**)&energy_dev, numBlocks * sizeof(double));
    hipMalloc((void**)&vir_dev, numBlocks * sizeof(double));
    hipMalloc((void**)&kind_dev, num * sizeof(int));
    hipMalloc((void**)&sigma_dev, num * sizeof(double));
    hipMalloc((void**)&epsilon_dev, num * sizeof(double));
    hipMalloc((void**)&rCut_dev, num * sizeof(double));

    // Copy arrays to the GPU
    hipMemcpy(r_dev, atoms[0]->r, 3 * num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(force_dev, atoms[0]->f, 3 * num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(kind_dev, atoms[0]->kind, num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(sigma_dev, atoms[0]->sigii, num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(epsilon_dev, atoms[0]->epsii, num * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(rCut_dev, atoms[0]->rCutii, num * sizeof(double), hipMemcpyHostToDevice);

    // Execute the kernel on the GPU
    setLJForceKernel<<<numBlocks, blockSize>>>(num, kind_dev, length, rCut_dev, sigma_dev,
                                               epsilon_dev, r_dev, force_dev, energy_dev, vir_dev);

    // Copy the 'energy' 'virial' and 'force' arrays back from the GPU to the CPU
    hipMemcpy(energy, energy_dev, numBlocks * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(vir, vir_dev, numBlocks * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(atoms[0]->f, force_dev, 3 * num * sizeof(double), hipMemcpyDeviceToHost);

    // Sum energy contributions of the CPU side
    potE = 0;
    virE = 0;
    for (int k = 0; k < numBlocks; k++) {
        potE += energy[k];
        virE += vir[k];
    }

    *potEnergy = potE;
    *virial = virE;

    // Free memory on the GPU side
    hipFree(energy_dev);
    hipFree(r_dev);
    hipFree(force_dev);
    hipFree(kind_dev);
    hipFree(sigma_dev);
    hipFree(epsilon_dev);
    hipFree(rCut_dev);

    // Free memory of the CPU side,
    free(energy);
    free(vir);
}

// Method: lrc
// Usage: n = lrc(num, nComp, volume, energyLRC, virialLRC);
// ---------------------------------------------------------
// ljLRC calculates the long range correction terms to both
// the energy and virial for an ensemble interacting via the
// lennard-Jones (12-6) potential.

void LJforce::lrc(int num, int* nComp, double boxV, double* energyLRC, double* virialLRC) {
    int i, j;
    const double PI = 3.141592654;
    double sigmaSq, sig3, rCutSq, rCut3, sig3R, sig9R, den;

    *energyLRC = 0.0;
    *virialLRC = 0.0;

    for (i = 0; i < num; i++)
        for (j = 0; j < num; j++) {
            sigmaSq = sigma[i][j] * sigma[i][j];
            rCutSq = rCut[i][j] * rCut[i][j];
            sig3 = sigma[i][j] * sigmaSq;
            rCut3 = rCut[i][j] * rCutSq;
            sig3R = sig3 / rCut3;
            sig9R = sig3R * sig3R * sig3R;
            den = nComp[i] * nComp[j] * sig3 / boxV;
            *energyLRC += (8.0 / 9.0) * den * PI * epsilon[i][j] * (sig9R - 3 * sig3R);
            *virialLRC += (16.0 / 9.0) * den * PI * epsilon[i][j] * (2 * sig9R - 3 * sig3R);
        }
}

// constructor
LJforce::LJforce(Atom** theAtoms) : Force(theAtoms) {
    epsilon = atoms[0]->getEpsilon();
    sigma = atoms[0]->getSigma();
    rCut = atoms[0]->getrCutOff();
}
