// File: ljenergyMC.cpp 
// -------------------- 
// File containing functions to implement the LJenergy
// class.

// This code was specifically developed to illustrate concepts in the accompanying book:
// R. J. Sadus, "Molecular Simulation of Fluids: Theory, Algorithms, Object-Orientation,
// and Parallel Computing," 2nd Ed. (Elsevier, Amsterdam, 2023). It can be used freely for
// any not for profit purpose or academic research application. The code has been validated,
// but it would be nonetheless prudent to test it further before publishing any results.
// Check the book's website for any subsequent updates.

#include "cudaSimKernels.h"
#include "ljenergyMC.h"
#include <iostream>
#include <stdlib.h>

using namespace std;

// Method: getTrialPotE
// Usage: getTrialPotE();
// -------------------- 
// The ljenergy method uses a GPU kernel to calculates the energy experieced
// by a given atom (index) due to pairwise interatomic interaction.
// The energies are calculated using the Lennard-Jones (6-12)
// GPU kernel. This can be replaced by kernels for any other potential
// The potential is truncated at a distance of rCut and 
// long range corrections must be applied outside of the method
// to obtain the full contributions to the energy. 
 
double LJenergy::getTrialPotE(int num, double length, int index, Atom **atom)
{
  int    *kind_dev;                   // atom type
  double *ri, *ri_dev;                // position vecotors of atom i
  double *r_dev;                      // position vectors for all atoms
  double *energy, *energy_dev;        // potential energy; 
  double *position, potE;
  double *sigma_dev, *epsilon_dev;
  double *rCut_dev;

  // Distribute the calculation among the threads
  int blockSize = numThreadsInBlock;
  int numBlocks = (num + blockSize - 1)/blockSize;

  // Allocate memory on the CPU side
  ri = (double *) malloc(3*sizeof(double)); //ri[0], ri[1] and ri[2] are the x, y, z components
  energy = (double *) malloc(numBlocks*sizeof(double));

  // Properties of the index atom
  position  = atom[index]->getTrialPosition();
  ri[0]     = position[0];
  ri[1]     = position[1];
  ri[2]     = position[2];

  // Allocate memory on the GPU
  hipMalloc((void **) &ri_dev,      3*sizeof(double));
  hipMalloc((void **) &r_dev,       3*num*sizeof(double));
  hipMalloc((void **) &energy_dev,  numBlocks*sizeof(double));
  hipMalloc((void **) &kind_dev,    num*sizeof(int));
  hipMalloc((void **) &sigma_dev,   num*sizeof(double));
  hipMalloc((void **) &epsilon_dev, num*sizeof(double));
  hipMalloc((void **) &rCut_dev,    num*sizeof(double));

  // Copy arrays to the GPU
  hipMemcpy(ri_dev,     ri,              3*sizeof(double),     hipMemcpyHostToDevice);
  hipMemcpy(r_dev,      atoms[0]->r,     3*num*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(kind_dev,   atoms[0]->kind,  num*sizeof(int),      hipMemcpyHostToDevice);
  hipMemcpy(sigma_dev,  atoms[0]->sigii, num*sizeof(double),   hipMemcpyHostToDevice);
  hipMemcpy(epsilon_dev,atoms[0]->epsii, num*sizeof(double),   hipMemcpyHostToDevice);
  hipMemcpy(rCut_dev,   atoms[0]->rCutii,num*sizeof(double),   hipMemcpyHostToDevice);
 
  // Execute  kernel on the GPU to determine the energy of the index atom with all
  // other distinct pairs
  energyLJKernel <<<numBlocks, blockSize>>> (num, index, kind_dev, length, rCut_dev, 
                               sigma_dev, epsilon_dev, ri_dev, r_dev, energy_dev); 

  // Copy the 'energy' array back from the GPU to the CPU
  hipMemcpy(energy, energy_dev, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

  // Sum energy contributions of the CPU side
  potE = 0;
  for(int k = 0; k < numBlocks; k++)
    potE +=  energy[k]; 

  // Free memory on the GPU side
  hipFree(energy_dev);
  hipFree(ri_dev);
  hipFree(r_dev);
  hipFree(kind_dev);
  hipFree(sigma_dev);
  hipFree(epsilon_dev);
  hipFree(rCut_dev);

  // Free memory of the CPU side
  free(energy);
  free(ri);

  return potE;
    
}

// Method: setEnergy
// Usage: setEnergy();
// -------------------- 
// The ljenergy method  uses a GPU kernel to calculates the energy experieced
// by all num atoms due to pairwise interatomic interaction.
// The forces are calculated using thekernel for the Lennard-Jones (6-12)
// This can be replaced by kernels for any other potential.
// The potential is truncated at a distance of rCut and long range
// corrections must be applied outsid e the method to obtain the
// the full contributions to the energy. 
 
void LJenergy::setEnergy(int num, int numComp, double length, 
                double *potEnergy)
{
  int    *kind_dev;                            //atom type;
  double *r_dev;                               // atom position vectors
  double *energy, *energy_dev;                 // potential energy; 
  double *sigma_dev, *epsilon_dev, *rCut_dev;
  double  potE;

  // Distribute the calculations among the threads  
  int blockSize = numThreadsInBlock;
  int numBlocks = (num + blockSize - 1)/blockSize;

  // Allocate memory on the CPU side
  energy = (double *) malloc(numBlocks*sizeof(double));

  // Allocate memory on the GPU
  hipMalloc((void **) &r_dev,       3*num*sizeof(double));
  hipMalloc((void **) &energy_dev,  numBlocks*sizeof(double));
  hipMalloc((void **) &kind_dev,    num*sizeof(int));
  hipMalloc((void **) &sigma_dev,   num*sizeof(double));
  hipMalloc((void **) &epsilon_dev, num*sizeof(double));
  hipMalloc((void **) &rCut_dev,    num*sizeof(double));

  // Copy arrays to the GPU
  hipMemcpy(r_dev,      atoms[0]->r,     3*num*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(kind_dev,   atoms[0]->kind,  num*sizeof(int),      hipMemcpyHostToDevice);
  hipMemcpy(sigma_dev,  atoms[0]->sigii, num*sizeof(double),   hipMemcpyHostToDevice);
  hipMemcpy(epsilon_dev,atoms[0]->epsii, num*sizeof(double),   hipMemcpyHostToDevice);
  hipMemcpy(rCut_dev,   atoms[0]->rCutii,num*sizeof(double),   hipMemcpyHostToDevice);

  // Execute the kernel on the GPU
  setLJEnergyKernel <<<numBlocks, blockSize>>> (num, kind_dev, length, 
                    rCut_dev, sigma_dev, epsilon_dev, r_dev, energy_dev); 

  // Copy the 'energy' array back from the GPU to the CPU
  hipMemcpy(energy, energy_dev, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

  // Sum energy contributions of the CPU side
  potE = 0;
  for(int k = 0; k < numBlocks; k++)
    potE +=  energy[k]; 

  *potEnergy = potE;

  // Free memory on the GPU side
  hipFree(energy_dev);
  hipFree(r_dev);
  hipFree(kind_dev);
  hipFree(sigma_dev);
  hipFree(epsilon_dev);
  hipFree(rCut_dev);

  // Free memory of the CPU side
  free(energy);

}

// Method: lrc
// Usage: n = lrc(num, nComp, volume, energyLRC);
// --------------------------------------------------------- 
// ljLRC calculates the long range correction terms to both
// the energy  for an ensemble interacting via the
// lennard-Jones (12-6) potential.

void LJenergy::lrc(int num, int *nComp, double boxV,
		double *energyLRC)
{
  int i, j;
  const double PI = 3.141592654;
  double eps, sigmaSq, sig3, rCutSq, rCut3, sig3R, sig9R, den;

  *energyLRC = 0.0;

  for(i = 0; i < num; i++)
      for(j = 0; j < num; j++){
          eps       = epsilon[i][j];
          sigmaSq   = sigma[i][j] * sigma[i][j];
          rCutSq    = rCut[i][j] * rCut[i][j];
          sig3      = sigma[i][j] * sigmaSq;
          rCut3     = rCut[i][j] * rCutSq;
          sig3R     = sig3/rCut3;
          sig9R     = sig3R * sig3R * sig3R;
          den       = nComp[i] * nComp[j] * sig3/boxV;
          *energyLRC += (8.0/9.0) * den * PI * eps
                    * (sig9R - 3 * sig3R);
      }
}

// constructor
LJenergy::LJenergy(Atom **theAtoms)
:Energy(theAtoms)
{
  epsilon = atoms[0]->getEpsilon();
  sigma   = atoms[0]->getSigma();
  rCut    = atoms[0]->getrCutOff();
}
