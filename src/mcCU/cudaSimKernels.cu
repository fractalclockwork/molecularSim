#include "hip/hip_runtime.h"
// File:  cudaSimKernals.cu
// -----------------
// This file contains the implementation details for the CUDA
// kernals required for molecular simulation using GPUs.

// This code was specifically developed to illustrate concepts in the accompanying book:
// R. J. Sadus, "Molecular Simulation of Fluids: Theory, Algorithms, Object-Orientation,
// and Parallel Computing," 2nd Ed. (Elsevier, Amsterdam, 2023). It can be used freely for
// any not for profit purpose or academic research application. The code has been validated,
// but it would be nonetheless prudent to test it further before publishing any results.
// Check the book's website for any subsequent updates.

#include "cudaSimKernels.h"

// Kernel to determine the LJ potential energy experienced by an index atom due
// to interations with all other distinct pairs of atoms. This kernel is used in
// MC simulations. It can be used for either mixtures or pure fluids.

__global__ void energyLJKernel(int num, int index, int* type, double length, double* rCut,
                               double* sigma, double* epsilon, double* ri, double* r,
                               double* potEnergy) {
    int k, nInt, kindi, kindj;
    double rXij, rYij, rZij;        // pair seperation vectors
    double rijSq, rCutSq;           // interatomic seperation squared
    double sigmaSq;                 // squared sigma values
    double sigma2, sigma6, sigma12; // multiples of LJ sigma
    double pot, potE;               // potential between 2 atoms
    double div, cut, eps, sig;

    __shared__ double kernelCache[numThreadsInBlock];

    int tID = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    int step = blockDim.x * gridDim.x;

    // Calculate energy experienced by the sellected atom (index)
    // with all other distinct pairs

    kindi = type[index];

    // Loop over all the other atoms
    potE = 0.0;
    for (int j = tID; j < num; j += step) {
        k = 3 * j;

        // Exclude self-interaction
        if (j != index) {
            rXij = ri[0] - r[k];
            rYij = ri[1] - r[k + 1];
            rZij = ri[2] - r[k + 2];

            // Apply periodic boundary conditions
            div = rXij / length;
            if (rXij >= 0)
                nInt = (int)(div + 0.5);
            else
                nInt = -(int)(0.5 - div);

            rXij -= length * nInt;

            div = rYij / length;
            if (rYij >= 0)
                nInt = (int)(div + 0.5);
            else
                nInt = -(int)(0.5 - div);

            rYij -= length * nInt;

            div = rZij / length;
            if (rZij >= 0)
                nInt = (int)(div + 0.5);
            else
                nInt = -(int)(0.5 - div);

            rZij -= length * nInt;

            rijSq = rXij * rXij + rYij * rYij + rZij * rZij;

            kindj = type[j];

            // Account for dissimilar atoms
            if (kindi == kindj)
                rCutSq = rCut[index] * rCut[index];
            else {
                cut = (rCut[index] + rCut[j]) / 2;
                rCutSq = cut * cut;
            }

            // Calculate forces for seprations
            // below the cutoff
            if (rijSq <= rCutSq) {
                // Accound for dissimilar atoms
                if (kindi == kindj) {
                    eps = epsilon[index];
                    sig = sigma[index];
                } else {
                    eps = sqrt(epsilon[index] * epsilon[j]);
                    sig = (sigma[index] + sigma[j]) / 2;
                }

                // Determine potential between i and j
                sigmaSq = sig * sig;
                sigma2 = sigmaSq / rijSq;
                sigma6 = sigma2 * sigma2 * sigma2;
                sigma12 = sigma6 * sigma6;
                pot = sigma12 - sigma6;
                potE += 4 * eps * pot;
            }
        }
    }
    // Set the kernel cache values
    kernelCache[cacheIndex] = potE;

    // Synchronize thread in this block
    __syncthreads();

    // Perform reductions, assuming numThreadsInBox is a power of 2.

    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i)
            kernelCache[cacheIndex] += kernelCache[cacheIndex + i];

        __syncthreads();

        i /= 2;
    }

    if (cacheIndex == 0)
        potEnergy[blockIdx.x] = kernelCache[0];
}

// Kernel to determine the complete LJ energy for all pairs of atoms. This kernel
// is used for MC simulations using CPUs. It can be used for either mixtures or
// pure fluids..

__global__ void setLJEnergyKernel(int num, int* type, double length, double* rCut, double* sigma,
                                  double* epsilon, double* r, double* energy) {
    int i, j, kindi, kindj, nInt;
    double rXi, rYi, rZi;           // position vectors for atom i
    double rXj, rYj, rZj;           // position vectors for atom j
    double rXij, rYij, rZij;        // pair seperation vectors
    double rijSq, rCutSq;           // interatomic seperation squared
    double sigmaSq;                 // squared sigma value
    double sigma2, sigma6, sigma12; // multiples of LJ sigma
    double pot, potE;               // potential between 2 atoms
    double div, cut, sig, eps;

    __shared__ double kernelECache[numThreadsInBlock];

    int tID = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    int step = blockDim.x * gridDim.x;

    // perform full N^2 calculation at the begining of the
    // simulation.

    // Calculate energy experienced by atoms due
    // interatomic pair interactions.

    potE = 0.0;

    for (i = tID; i < num; i += step) {
        // Component type
        kindi = type[i];

        // Select position vectors of atom i
        rXi = r[3 * i];
        rYi = r[3 * i + 1];
        rZi = r[3 * i + 2];

        for (j = 0; j < num; j++) {
            if (j != i) {
                // Component type
                kindj = type[j];

                // Select position vectors
                rXj = r[3 * j];
                rYj = r[3 * j + 1];
                rZj = r[3 * j + 2];

                // Calculate pair seperation
                rXij = rXi - rXj;
                rYij = rYi - rYj;
                rZij = rZi - rZj;

                // Apply periodic boundary conditions
                div = rXij / length;
                if (rXij >= 0)
                    nInt = (int)(div + 0.5);
                else
                    nInt = -(int)(0.5 - div);

                rXij -= length * nInt;

                div = rYij / length;
                if (rYij >= 0)
                    nInt = (int)(div + 0.5);
                else
                    nInt = -(int)(0.5 - div);

                rYij -= length * nInt;

                div = rZij / length;
                if (rZij >= 0)
                    nInt = (int)(div + 0.5);
                else
                    nInt = -(int)(0.5 - div);

                rZij -= length * nInt;

                rijSq = rXij * rXij + rYij * rYij + rZij * rZij;

                // Account for dissimilar atoms

                if (kindi == kindj)
                    rCutSq = rCut[i] * rCut[i];
                else {
                    cut = (rCut[i] + rCut[j]) / 2;
                    rCutSq = cut * cut;
                }

                // Calculate forces for seprations
                // below the cutoff

                if (rijSq <= rCutSq) {

                    // Account for dissimilar atoms
                    if (kindi == kindj) {
                        eps = epsilon[i];
                        sig = sigma[i];
                    } else {
                        eps = sqrt(epsilon[i] * epsilon[j]);
                        sig = (sigma[i] + sigma[j]) / 2;
                    }

                    // Determine potential between i and j
                    sigmaSq = sig * sig;
                    sigma2 = sigmaSq / rijSq;
                    sigma6 = sigma2 * sigma2 * sigma2;
                    sigma12 = sigma6 * sigma6;
                    pot = sigma12 - sigma6;
                    potE += 4 * eps * pot;
                }
            }
        }
    }

    // Set the kernel cache values
    kernelECache[cacheIndex] = potE / 2;

    // Synchronize thread in this block
    __syncthreads();

    // Perform reductions, assuming numThreadsInBox is a power of 2.

    int k = blockDim.x / 2;
    while (k != 0) {
        if (cacheIndex < k)
            kernelECache[cacheIndex] += kernelECache[cacheIndex + k];

        __syncthreads();

        k /= 2;
    }

    if (cacheIndex == 0)
        energy[blockIdx.x] = kernelECache[0];
}

__global__ void cudaReduce(double* sum, double* array) {
    extern __shared__ double temp[];

    int tID = threadIdx.x;

    temp[tID] = array[tID + blockIdx.x * blockDim.x];

    for (int i = blockDim.x >> 1; i >= 1; i >>= 1) {
        __syncthreads();

        if (tID < i)
            temp[tID] += temp[tID + i];
    }

    if (tID == 0)
        sum[blockIdx.x] = temp[0];
}

// Kernal to reduce threads is adapted from M. Harris (NVIDIA)
__global__ void cudaReduceThread(double* globalOut, double* globalIn) {
    extern __shared__ double sharedData[];

    // for each thread, load one element from global to shared memory
    unsigned int tID = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    sharedData[tID] = globalIn[i];
    __syncthreads();

    // perform reduction on shared memory assuming threads are powers of 2
    for (unsigned int j = 1; j < blockDim.x; j *= 2) {
        if (tID % (2 * j) == 0)
            sharedData[tID] += sharedData[tID + j];

        __syncthreads();
    }

    // write result for the block to global memory
    if (tID == 0)
        globalOut[blockIdx.x] = sharedData[0];
}
